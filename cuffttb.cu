#include<stdio.h>
#include<math.h>
#include<complex.h>
#include<hipfft/hipfft.h>
#define Nx 10
#define BATCH 1
typedef double complex cplx;

int main()
{
	hipfftHandle plan;
	hipDoubleComplex *data;
	hipDoubleComplex dataH[Nx];
	for(int i=0;i<Nx;i++)
	{
		dataH[i].x=i;
		dataH[i].y=0.0;
	}
	hipMalloc((void**)&data,sizeof(hipDoubleComplex)*Nx*BATCH);
	if (hipGetLastError() != hipSuccess)
	{
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return 0;	
	}
	if (hipfftPlan1d(&plan, Nx, HIPFFT_Z2Z,BATCH) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return 0;	
	}	
	size_t sizeData= Nx*sizeof(hipDoubleComplex);
	hipMemcpy(data,dataH,sizeData,hipMemcpyHostToDevice);
	if (hipfftExecZ2Z(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
	{
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
		return 0;
	}
	hipMemcpy(dataH,data,sizeData,hipMemcpyDeviceToHost);
	printf("\n");
	for(int i=0;i<Nx;i++)
		printf("%f:%f ",dataH[i].x,dataH[i].y);
	return 0;
}
